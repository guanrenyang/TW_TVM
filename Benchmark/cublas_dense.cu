#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i)) // macro to index a 2D array stored in row-major order





void matrixMul(const float *A, const float *B, float *C, const int m, const int n, const int k) {
    // Allocate device memory for matrices A, B, and C
    float *d_A, *d_B, *d_C;
    hipError_t cudaStatus;
    cudaStatus = hipMalloc((void**)&d_A, m*k*sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }
    cudaStatus = hipMalloc((void**)&d_B, k*n*sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_A);
        return;
    }
    cudaStatus = hipMalloc((void**)&d_C, m*n*sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_A);
        hipFree(d_B);
        return;
    }

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&handle);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS initialization failed\n");
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    // Copy matrices A and B from host to device
    hipblasStatus_t = hipblasSetMatrix(m, k, sizeof(float), A, m, d_A, m);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS set matrix A failed\n");
        hipblasDestroy(handle);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }
    hipblasStatus_t = hipblasSetMatrix(k, n, sizeof(float), B, k, d_B, k);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS set matrix B failed\n");
        hipblasDestroy(handle);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    // Allocate CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // Compute C = alpha*A*B + beta*C using cuBLAS
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasStatus_t = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS matrix multiplication failed\n");
        hipblasDestroy(handle);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    // Record stop event and wait for completion
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed time: %f ms\n", elapsedTime);
 
    // Copy matrix C from device to host
    hipblasStatus_t = hipblasGetMatrix(m, n, sizeof(float), d_C, m, C, m);
    if (hipblasStatus_t != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS get matrix C failed\n");
        hipblasDestroy(handle);
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return;
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Destroy cuBLAS handle
    hipblasDestroy(handle);

}


void matrixMul_cpu(const float *A, const float *B, float *C, const int m, const int n, const int k) {
    // Perform matrix multiplication C = A*B
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            float sum = 0.0f;
            for (int l = 0; l < k; ++l) {
                sum += A[i*k+l] * B[l*n+j];
            }
            C[i*n+j] = sum;
        }
    }
}

int main() {
    const int m = 1024; // number of rows in matrix A and matrix C
    const int n = 1024; // number of columns in matrix B and matrix C
    const int k = 1024; // number of columns in matrix A and number of rows in matrix B

    // Allocate host memory for matrices A, B, and C
    float *h_A = (float*)malloc(m*k*sizeof(float));
    float *h_B = (float*)malloc(k*n*sizeof(float));
    float *h_C = (float*)malloc(m*n*sizeof(float));
    float *h_C_verify = (float*)malloc(m*n*sizeof(float));

    // Initialize matrices A and B with random values
    for (int i = 0; i < m*k; ++i) {
        h_A[i] = ((float) rand()) / ((float) RAND_MAX);
    }
    for (int i = 0; i < k*n; ++i) {
        h_B[i] = ((float) rand()) / ((float) RAND_MAX);
    }

    // Compute matrix C = A*B using GPU kernel
    matrixMul(h_A, h_B, h_C, m, n, k);
    matrixMul(h_A, h_B, h_C, m, n, k);
    matrixMul(h_A, h_B, h_C, m, n, k);
    matrixMul(h_A, h_B, h_C, m, n, k);
    // matrixMul_cpu(h_A, h_B, h_C_verify, m, n, k);

    // // Print matrix C
    // printf("Matrix C =\n");
    // for (int i = 0; i < m; ++i) {
    //     for (int j = 0; j < n; ++j) {
    //         printf("%f ", h_C[IDX2C(i,j,m)]);
    //     }
    //     printf("\n");
    // }



    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_verify);
    return 0;
}

